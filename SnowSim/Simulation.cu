#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <iostream>

// TMP
#include "svd_test.h"
#include "float3x3.h"

#include "Serializable.h"
#include "Simulation.cuh"
#include "Cube.h"
#include "Sphere.h"
#include "Plane.h"

#define NUM_THREADS 128

#define GRAVITY make_float3(0.f, -9.8f, 0.f)

#define ALPHA .05f

__device__
void atomicAdd(float3* address, float3 val)
{
	atomicAdd(&address->x, val.x);
	atomicAdd(&address->y, val.y);
	atomicAdd(&address->z, val.z);
}

__device__
float3 sign(float3 a)
{
	float3 val;
	
	return make_float3(
		sign(a.x),
		sign(a.y),
		sign(a.z));
}

__host__ __device__
float bSplineFalloff(float d)
{
	float _d =
		((0 <= d && d < 1) * (.5*d*d*d - d * d + 2.f / 3.f) +
		(1 <= d && d < 2) * (-1.f / 6.f*d*d*d + d * d - 2 * d + 4.f / 3.f));

	return fabs(_d);
}
/*
__device__
float bSplineFalloff(float w)
{
	float a =
		(0.f <= w && w < 1.f) *
		(.5f * w * w * w -
			w * w +
			2.f / 3.f);
	float b =
		(1.f <= w && w < 2.f) *
		(-1.f / 6.f * w * w * w +
			w * w -
			2.f * w +
			4.f / 3.f);

	return a + b;
}*/

__device__
float bSplineGradFalloff(float w)
{
	float a =
		(0.f <= w && w < 1.f) *
		(1.5f * w * w - 2.f * w);
	float b =
		(1.f <= w && w < 2.f) *
		(-.5f * w * w + 2.f * w - 2.f);

	return a + b;
}

__device__
void computeWeightAndGrad(
	const float3& val,
	const float3& val2,
	float& weight,
	float3& weightGrad)
{
	// TODO: rename shit
	const float3 N =
		make_float3(
			bSplineFalloff(val2.x),
			bSplineFalloff(val2.y),
			bSplineFalloff(val2.z));

	weight =
		N.x *
		N.y *
		N.z;

	const float3 Nx =
		val *
		make_float3(
			bSplineGradFalloff(val2.x),
			bSplineGradFalloff(val2.y),
			bSplineGradFalloff(val2.z));
	weightGrad.x = Nx.x * N.y * N.z;
	weightGrad.y = N.x * Nx.y * N.z;
	weightGrad.z = N.x * N.y * Nx.z;
}

__device__
void computeWeightAndGrad(
	const float3& val,
	float& weight,
	float3& weightGrad)
{
	// TODO: rename shit
	const float3 sdx = sign(val);
	const float3 adx = fabs(val);
	const float3 N =
		make_float3(
			bSplineFalloff(adx.x),
			bSplineFalloff(adx.y),
			bSplineFalloff(adx.z));

	weight =
		N.x *
		N.y *
		N.z;

	const float3 Nx =
		sdx *
		make_float3(
			bSplineGradFalloff(adx.x),
			bSplineGradFalloff(adx.y),
			bSplineGradFalloff(adx.z));
	weightGrad.x = Nx.x * N.y * N.z;
	weightGrad.y = N.x * Nx.y * N.z;
	weightGrad.z = N.x * N.y * Nx.z;
}

__global__
void InitMass(
	SnowParticle* particles,
	float* voxelMass,
	GridInfo gridInfo,
	uint numParticles)
{
	int id = 
		blockIdx.y * gridDim.x * blockDim.x +
		blockIdx.x * blockDim.x + threadIdx.x;

	// Bounds check.
	if (numParticles <= id)
	{
		return;
	}

	const SnowParticle& particle =
		particles[id];

	float3 cellIndexF =
		gridInfo.GetCellPosF(
			particles[id].position);

	uint3 relPos =
		GridInfo::GetRelativePos(
			threadIdx.y,
			make_uint3(4, 4, 4));

	int3 cell =
		make_int3(
			cellIndexF.x - 1,
			cellIndexF.y - 1,
			cellIndexF.z - 1);
	cell += make_int3(
		relPos.x,
		relPos.y,
		relPos.z);

	// Check within grid.
	if (gridInfo.InsideGrid(cell))
	{
		float3 delta =
			fabs(make_float3(
				cell.x,
				cell.y,
				cell.z)
				- cellIndexF);

		// 1D b-spline falloff.
		float weight =
			bSplineFalloff(delta.x) *
			bSplineFalloff(delta.y) *
			bSplineFalloff(delta.z);

		uint3 cellU =
			make_uint3(
				cell.x,
				cell.y,
				cell.z);

		atomicAdd(
			&voxelMass[GridInfo::GetIndex(
				cellU, make_uint3(
					gridInfo.width + 1,
					gridInfo.height + 1,
					gridInfo.depth + 1))],
			particle.mass * weight);
	}
}

__global__
void InitDensity(
	SnowParticle* particles,
	float* voxelMass,
	GridInfo gridInfo,
	uint numParticles)
{
	int id =
		blockIdx.y * gridDim.x * blockDim.x +
		blockIdx.x * blockDim.x + threadIdx.x;

	// Bounds check.
	if (numParticles <= id)
	{
		return;
	}

	SnowParticle& particle =
		particles[id];

	float3 cellIndexF =
		gridInfo.GetCellPosF(
			particles[id].position);

	uint3 relPos =
		GridInfo::GetRelativePos(
			threadIdx.y,
			make_uint3(4, 4, 4));

	int3 cell =
		make_int3(
			cellIndexF.x - 1,
			cellIndexF.y - 1,
			cellIndexF.z - 1);
	cell += make_int3(
		relPos.x,
		relPos.y,
		relPos.z);

	// Check within grid.
	if (gridInfo.InsideGrid(cell))
	{
		float3 delta =
			fabs(make_float3(
				cell.x,
				cell.y,
				cell.z)
				- cellIndexF);

		// 1D b-spline falloff.
		float weight =
			bSplineFalloff(delta.x) *
			bSplineFalloff(delta.y) *
			bSplineFalloff(delta.z);

		uint3 cellU =
			make_uint3(
				cell.x,
				cell.y,
				cell.z);

		atomicAdd(
			&particle.volume,
				voxelMass[GridInfo::GetIndex(
					cellU, make_uint3(
						gridInfo.width + 1,
						gridInfo.height + 1,
						gridInfo.depth + 1))] *
				weight /
				(gridInfo.scale * gridInfo.scale * gridInfo.scale));
	}
}

__global__
void InitVolume(
	SnowParticle* particles,
	GridInfo gridInfo,
	uint numParticles)
{
	uint id = blockIdx.x * blockDim.x + threadIdx.x;

	// Bounds check.
	if (numParticles <= id)
	{
		return;
	}

	SnowParticle& particle =
		particles[id];
	
	particle.volume =
		particle.mass / particle.volume;
}

void Simulation::SetupSim(
	Grid<GridCell>* grid,
	SnowParticle* particleList,
	uint numParticles,
	Obstacle* obstacles,
	uint numObstacles)
{
	{
		this->grid = grid;
		this->particles = particleList;
		this->numParticles = numParticles;
		this->obstacles = obstacles;
		this->numObstacles = numObstacles;
	}

	GridInfo gridInfo =
		*grid->GetGridInfo();

	size_t numNodes =
		(gridInfo.width + 1) *
		(gridInfo.height + 1) *
		(gridInfo.depth + 1);

	float* voxelMass;
	{
		hipError_t(
			hipMalloc(
				&voxelMass,
				numNodes * sizeof(float)));
		hipError_t(
			hipMemset(
				voxelMass,
				0,
				numNodes * sizeof(float)));
	}

	const dim3 threads(
		NUM_THREADS / 64,
		64,
		1);
	const dim3 blocks(
		(numParticles + NUM_THREADS - 1) / NUM_THREADS,
		64,
		1);

	const dim3 threads_2(
		NUM_THREADS,
		1,
		1);
	const dim3 blocks_2(
		(numParticles + NUM_THREADS - 1) / NUM_THREADS,
		1,
		1);

	{
		InitMass<<<blocks, threads>>>(
			particleList,
			voxelMass,
			gridInfo,
			numParticles);

		InitDensity<<<blocks, threads>>>(
			particleList,
			voxelMass,
			gridInfo,
			numParticles);

		InitVolume<<<blocks_2, threads_2>>> (
			particleList,
			gridInfo,
			numParticles);
	}

	{
		hipError_t(
			hipFree(voxelMass));
	}
}

__global__
void UpdateObstacles(
	Obstacle* obstacles,
	const uint colliderNum,
	float deltaT)
{
	uint id = blockIdx.x * blockDim.x + threadIdx.x;

	if (colliderNum <= id)
	{
		return;
	}

	Obstacle& obstacle =
		obstacles[id];

	obstacle.pos +=
		obstacle.vel *
		deltaT;
}

__global__
void SolveSystem(
	SnowParticle* particles,
	SnowParticleExternalData* externalData,
	uint numParticles)
{
	uint id = blockIdx.x * blockDim.x + threadIdx.x;

	if (numParticles <= id)
	{
		return;
	}
	
	const SnowParticle& particle =
		particles[id];

	float3x3& plasticity =
		particles[id].plasticity;
	float3x3& elasticity =
		particles[id].elasticity;

	float detP =
		plasticity.det();
	float detE =
		elasticity.det();
	
	float3x3 pD =
		elasticity.polarDecomp();
	
	const Mat& material =
		particle.material;

	float muComp = material.mu *
		expf(material.xi * (1.f - detP));
	float lambdaComp = material.lambda *
		expf(material.xi * (1.f - detP));

	externalData[id].sigma =
		(2.f * muComp * (elasticity - pD).multABt(elasticity) +
		float3x3(detE * lambdaComp * (detE - 1.f))) * -particle.volume;
}

__global__
void ComputeSim(
	SnowParticle* particles,
	SnowParticleExternalData* externalData,
	GridCell* gridCell,
	GridInfo gridInfo,
	uint numParticles)
{
	int id =
		blockIdx.y * gridDim.x * blockDim.x +
		blockIdx.x * blockDim.x + threadIdx.x;

	// Bounds check.
	if (numParticles <= id)
	{
		return;
	}

	const SnowParticle& particle =
		particles[id];

	float3 cellIndexF =
		gridInfo.GetCellPosF(
			particles[id].position);

	uint3 relPos =
		GridInfo::GetRelativePos(
			threadIdx.y,
			make_uint3(4, 4, 4));

	int3 cell =
		make_int3(
			cellIndexF.x - 1,
			cellIndexF.y - 1,
			cellIndexF.z - 1);
	cell += make_int3(
		relPos.x,
		relPos.y,
		relPos.z);

	// Check within grid.
	if (gridInfo.InsideGrid(cell))
	{
		float3 delta =
			cellIndexF -
			make_float3(
				cell.x,
				cell.y,
				cell.z);

		uint3 cellU =
			make_uint3(
				cell.x,
				cell.y,
				cell.z);
		
		GridCell& voxel =
			gridCell[GridInfo::GetIndex(
				cellU, make_uint3(
					gridInfo.width + 1,
					gridInfo.height + 1,
					gridInfo.depth + 1))];
		
		float weight;
		float3 weightGrad;
		computeWeightAndGrad(
			delta,
			weight,
			weightGrad);

		atomicAdd(
			&voxel.mass,
			particle.mass * weight);
		atomicAdd(
			&voxel.velocity,
			particle.velocity * particle.mass * weight);
		atomicAdd(
			&voxel.force,
			externalData[id].sigma * weightGrad);
	}
}

__device__
void ProcessObstacles(
	const Obstacle* obstacles,
	const uint numObstacles,
	const float3& pos,
	float3& velocity)
{
	for (uint i = 0; i < numObstacles; i++)
	{
		const Obstacle& obstacle =
			obstacles[i];

		if (computeHit(obstacle, pos))
		{
			float3 vDelta =
				velocity - obstacle.vel;

			float3 normal =
				computeNormal(obstacle, pos);

			float angle =
				dot(vDelta, normal);

			// Objects moving into each other.
			if (angle < 0.f)
			{
				float3 ref =
					vDelta - normal * angle;
				float magnitude =
					length(ref);

				if (magnitude <= -obstacle.friction * angle)
				{
					vDelta = make_float3(
						0.f,
						0.f,
						0.f);
				}
				else
				{
					vDelta =
						(1.f + obstacle.friction * angle / magnitude) *
						ref;
				}
			}

			velocity =
				vDelta +
				obstacle.vel;
		}
	}
}

__global__
void ComputeCellVel(
	bool updateDeltaV,
	SnowParticle* particles,
	SnowParticleExternalData* externalData,
	GridCell* gridCell,
	GridInfo gridInfo,
	float deltaT,
	uint numParticles,
	Obstacle* obstacles,
	uint numObstacles)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	// Bounds check.
	if ((gridInfo.width + 1) *
		(gridInfo.height + 1) *
		(gridInfo.depth + 1) <= id)
	{
		return;
	}

	GridCell& voxel =
		gridCell[id];

	if (0.f < voxel.mass)
	{
		float inv =
			1.f / voxel.mass;

		voxel.velocity *=
			inv;
		voxel.deltaV =
			voxel.velocity;
		voxel.force +=
			make_float3(
				voxel.mass * GRAVITY.x,
				voxel.mass * GRAVITY.y,
				voxel.mass * GRAVITY.z);
		voxel.velocity +=
			deltaT *
			inv *
			voxel.force;

		// Collisions.
		{
			uint3 relPos =
				GridInfo::GetRelativePos(
					id,
					make_uint3(
						gridInfo.width + 1,
						gridInfo.height + 1,
						gridInfo.depth + 1));

			float3 pos =
				make_float3(
					relPos.x,
					relPos.y,
					relPos.z) * gridInfo.scale +
					gridInfo.position;

			ProcessObstacles(
				obstacles,
				numObstacles,
				pos,
				voxel.velocity);
		}

		if (updateDeltaV)
		{
			voxel.deltaV =
				voxel.velocity - voxel.deltaV;
		}
	}
}

__device__
float3x3 ComputeVelGrad(
	SnowParticle& particle,
	GridCell* gridCell,
	GridInfo gridInfo)
{
	float3 cellF =
		gridInfo.GetCellPosF(particle.position);

	// Bounds for looping.
	uint3 minIndx, maxIndx;
	{
		minIndx = clamp(
			make_uint3(
				ceilf(cellF.x),
				ceilf(cellF.y),
				ceilf(cellF.z))
			- make_uint3(2, 2, 2),
			make_uint3(0, 0, 0),
			make_uint3(
				gridInfo.width,
				gridInfo.height,
				gridInfo.depth));
		maxIndx = clamp(
			make_uint3(
				floorf(cellF.x),
				floorf(cellF.y),
				floorf(cellF.z))
			+ make_uint3(2, 2, 2),
			make_uint3(0, 0, 0),
			make_uint3(
				gridInfo.width,
				gridInfo.height,
				gridInfo.depth));
	}

	// PIC / FLIP sim.
	float3 pic, flip;
	float3x3 velGrad(0.f);
	{
		pic = flip = make_float3(0.f, 0.f, 0.f);

		for (uint x = minIndx.x; x <= maxIndx.x; x++)
		{
			float3 data, s;

			for (uint y = minIndx.y; y <= maxIndx.y; y++)
			{
				for (uint z = minIndx.z; z <= maxIndx.z; z++)
				{
					{
						data = cellF - make_float3(x, y, z);
						s = sign(data);

						// Abs.
						data *= s;
					}
					
					float weight;
					float3 wGrad;
					{
						computeWeightAndGrad(
							s,
							data,
							weight,
							wGrad);
					}

					const GridCell& voxel =
						gridCell[
							GridInfo::GetIndex(
								make_uint3(x, y, z),
								make_uint3(
									gridInfo.width + 1,
									gridInfo.height + 1,
									gridInfo.depth + 1))];
					
					velGrad =
						velGrad +
						float3x3::outerProduct(
							voxel.velocity,
							wGrad);
					/*
					velGrad.d[0] += voxel.velocity.x;
					velGrad.d[1] += voxel.velocity.y;
					velGrad.d[2] += voxel.velocity.z;*/


					pic += voxel.velocity * weight;
					flip += voxel.deltaV * weight;
				}
			}
		}
	}
	
	particle.velocity =
		lerp(pic, particle.velocity + flip, ALPHA);

	return velGrad;
}

__device__
void ComputeDeformGrad(
	SnowParticle& particle,
	float3x3 velGrad,
	float deltaT)
{
	particle.elasticity =
		(deltaT * velGrad + float3x3()) *
		particle.elasticity;
	
	const Mat& material =
		particle.material;

	float3x3 u, s, v;
	particle.elasticity.svdDecomp(
		u, s, v);
	
	float3x3 sClamp;
	{
		sClamp.d[0] = clamp(s.d[0],
			material.compressionRatio,
			material.stretchRatio);
		sClamp.d[4] = clamp(s.d[4],
			material.compressionRatio,
			material.stretchRatio);
		sClamp.d[8] = clamp(s.d[8],
			material.compressionRatio,
			material.stretchRatio);
	}

	float3x3 sClampInv;
	{
		sClampInv.d[0] = 1.f / sClamp.d[0];
		sClampInv.d[4] = 1.f / sClamp.d[4];
		sClampInv.d[8] = 1.f / sClamp.d[8];
	}
	
	particle.plasticity =
		v.multABCt(sClampInv, u) *
		particle.elasticity *
		particle.plasticity;
	particle.elasticity =
		u.multABCt(sClamp, v);
}

__global__
void UpdateParticles(
	SnowParticle* particles,
	SnowParticleExternalData* externalData,
	GridCell* gridCell,
	GridInfo gridInfo,
	float deltaT,
	uint numParticles,
	Obstacle* obstacles,
	const uint numObstacles)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	// Bounds check.
	if (numParticles <= id)
	{
		return;
	}

	SnowParticle& particle =
		particles[id];

	float3x3 velGrad(0.f);

	velGrad =
		ComputeVelGrad(
			particle,
			gridCell,
			gridInfo);

	ComputeDeformGrad(
		particle,
		velGrad,
		deltaT);
	
	ProcessObstacles(
		obstacles,
		numObstacles,
		particle.position,
		particle.velocity);

	particle.position +=
		deltaT * particle.velocity;
}

void Simulation::StepSim(
	float deltaT,
	uint frame)
{
	//std::cout << "FRAME " << frame << std::endl;

	// Clear data.
	{
		hipError_t(
			hipMemset(
				grid->Data(),
				0,
				(grid->GetWidth() + 1) *
				(grid->GetHeight() + 1) *
				(grid->GetDepth() + 1) *
				sizeof(GridCell)));
	}

	// Allocate cache.
	SnowParticleExternalData* externalData;
	{
		hipError_t(
			hipMalloc(
				&externalData,
				numParticles * sizeof(SnowParticleExternalData)));
	}

	{
		const dim3 threads(
			NUM_THREADS,
			1,
			1);
		const dim3 blocks(
			(numObstacles + NUM_THREADS - 1) / NUM_THREADS,
			1,
			1);

		UpdateObstacles<<<blocks, threads>>>(
			obstacles,
			numObstacles,
			deltaT);
	}

	{
		const dim3 threads(
			NUM_THREADS,
			1,
			1);
		const dim3 blocks(
			(numParticles + NUM_THREADS - 1) / NUM_THREADS,
			1,
			1);

		SolveSystem<<<blocks, threads>>>(
			particles,
			externalData,
			numParticles);

#ifdef CHECK
		{
			unsigned int size =
				numParticles;
			auto externalDataCPU =
				new SnowParticleExternalData[size];

			hipError_t(
				hipMemcpy(
					externalDataCPU,
					externalData,
					size * sizeof(SnowParticleExternalData),
					hipMemcpyDeviceToHost));

			float3* vals =
				new float3[size * 3];

			for (uint i = 0; i < size; i++)
			{
				float3 v;

				for (uint k = 0; k < 9; k+=3)
				{
					v.x =
						externalDataCPU[i].sigma.d[k];
					v.y =
						externalDataCPU[i].sigma.d[k + 1];
					v.z =
						externalDataCPU[i].sigma.d[k + 2];

					vals[i * 3 + k / 3] = v;
				}
			}

#ifndef _DEBUG
			Serializable::Store(
				vals,
				size * 3,
				"externalData.txt",
				frame);
#else
			Serializable::Compare(
				vals,
				"externalData.txt",
				frame);
#endif
			delete[] externalDataCPU;
			delete[] vals;
		}
#endif
	}
	/*
	std::cout << "2____________" << std::endl;
	{
		SnowParticleExternalData* externalDataCPU =
			new SnowParticleExternalData[numParticles];

		hipError_t(hipMemcpy(
			externalDataCPU,
			externalData,
			numParticles * sizeof(SnowParticleExternalData),
			hipMemcpyDeviceToHost));

		for (uint i = 0; i < numParticles; i++)
		{
			std::cout << i << std::endl;
			externalDataCPU[i].sigma.print();
		}
		system("PAUSE");
	}
	*/
	{
		const dim3 threads(
			NUM_THREADS / 64,
			64,
			1);
		const dim3 blocks(
			(numParticles + NUM_THREADS - 1) / NUM_THREADS,
			64,
			1);
		
		ComputeSim<<<blocks, threads>>> (
			particles,
			externalData,
			grid->Data(),
			*grid->GetGridInfo(),
			numParticles);

#ifdef CHECK
		{
			unsigned int size =
				(grid->GetWidth() + 1) *
				(grid->GetHeight() + 1) *
				(grid->GetDepth() + 1);
			auto cellCPU =
				new GridCell[size];

			hipError_t(
				hipMemcpy(
					cellCPU,
					grid->Data(),
					size * sizeof(GridCell),
					hipMemcpyDeviceToHost));

			float3* vals =
				new float3[size * 3];

			for (uint i = 0; i < size; i++)
			{
				vals[i * 3] =
					make_float3(
						cellCPU[i].mass,
						0,
						0);
				vals[i * 3 + 1] =
					cellCPU[i].velocity;
				vals[i * 3 + 2] =
					cellCPU[i].force;
			}

#ifndef _DEBUG
			Serializable::Store(
				vals,
				size * 3,
				"cellData.txt",
				frame);
#else
			Serializable::Compare(
				vals,
				"cellData.txt",
				frame);
#endif
			delete[] cellCPU;
			delete[] vals;
		}
#endif
	}

	/*
	std::cout << "3____________" << std::endl;
	{
		uint numNodes =
			(grid->GetWidth() + 1) *
			(grid->GetHeight() + 1) *
			(grid->GetDepth() + 1);

		GridCell* nodeCPU =
			new GridCell[numNodes];

		hipError_t(hipMemcpy(
			nodeCPU,
			grid->Data(),
			numNodes * sizeof(GridCell),
			hipMemcpyDeviceToHost));

		for (uint i = 0; i < numNodes; i++)
		{
			if (nodeCPU[i].mass != 0)
			{
				std::cout << i << " " <<
					nodeCPU[i].velocity.x << " " <<
					nodeCPU[i].velocity.y << " " <<
					nodeCPU[i].velocity.z << " ";
				std::cout << std::endl;
			}
		}
	}*/
	
	{
		const dim3 threads(
			NUM_THREADS,
			1,
			1);
		const dim3 blocks(
			((grid->GetWidth() + 1) *
				(grid->GetHeight() + 1) *
				(grid->GetDepth() + 1)
				+ NUM_THREADS - 1) / NUM_THREADS,
			1,
			1);
		
		ComputeCellVel<<<blocks, threads>>>(
			true,
			particles,
			externalData,
			grid->Data(),
			*grid->GetGridInfo(),
			deltaT,
			numParticles,
			obstacles,
			numObstacles);

#ifdef CHECK
		{
			unsigned int size =
				(grid->GetWidth() + 1) *
				(grid->GetHeight() + 1) *
				(grid->GetDepth() + 1);
			auto cellCPU =
				new GridCell[size];

			hipError_t(
				hipMemcpy(
					cellCPU,
					grid->Data(),
					size * sizeof(GridCell),
					hipMemcpyDeviceToHost));

			float3* vals =
				new float3[size * 3];

			for (uint i = 0; i < size; i++)
			{
				vals[i * 3] =
					cellCPU[i].velocity;
				vals[i * 3 + 1] =
					cellCPU[i].force;
				vals[i * 3 + 2] =
					cellCPU[i].deltaV;
			}

#ifndef _DEBUG
			Serializable::Store(
				vals,
				size * 3,
				"cellData2.txt",
				frame);
#else
			Serializable::Compare(
				vals,
				"cellData2.txt",
				frame);
#endif
			delete[] cellCPU;
			delete[] vals;
		}
#endif
	}

	/*
	std::cout << "4____________" << std::endl;
	{
		uint numNodes =
			(grid->GetWidth() + 1) *
			(grid->GetHeight() + 1) *
			(grid->GetDepth() + 1);

		GridCell* nodeCPU =
			new GridCell[numNodes];

		hipError_t(hipMemcpy(
			nodeCPU,
			grid->Data(),
			numNodes * sizeof(GridCell),
			hipMemcpyDeviceToHost));
		
		int m = 0;

		for (uint i = 0; i < numNodes; i++)
		{
			if (nodeCPU[i].mass != 0)
			{
				m++;
				std::cout << i << " " <<
					nodeCPU[i].mass << " " <<
					nodeCPU[i].velocity.x << " " <<
					nodeCPU[i].velocity.y << " " <<
					nodeCPU[i].velocity.z << " ";
				std::cout << std::endl;
			}
		}

		std::cout << m << std::endl;
	}*/
	
	// TODO: implicit.
	{

	}

	{
		const dim3 threads(
			NUM_THREADS,
			1,
			1);
		const dim3 blocks(
			(numParticles + NUM_THREADS - 1) / NUM_THREADS,
			1,
			1);
		
		UpdateParticles<<<blocks, threads>>>(
			particles,
			externalData,
			grid->Data(),
			*grid->GetGridInfo(),
			deltaT,
			numParticles,
			obstacles,
			numObstacles);


#ifdef CHECK
		{
			unsigned int size =
				numParticles;
			auto particleCPU =
				new SnowParticle[size];

			hipError_t(
				hipMemcpy(
					particleCPU,
					particles,
					size * sizeof(SnowParticle),
					hipMemcpyDeviceToHost));

			float3* vals =
				new float3[size * 8];

			for (uint i = 0; i < size; i++)
			{
				auto& p =
					particleCPU[i];

				vals[i * 8] = p.position;
				vals[i * 8 + 1] = p.velocity;

				float3 v, ev;

				for (uint k = 0; k < 9; k += 3)
				{
					v.x =
						p.elasticity.d[k];
					v.y =
						p.elasticity.d[k + 1];
					v.z =
						p.elasticity.d[k + 2];

					ev.x =
						p.plasticity.d[k];
					ev.y =
						p.plasticity.d[k + 1];
					ev.z =
						p.plasticity.d[k + 2];

					vals[i * 8 + 2 + k / 3] = v;
					vals[i * 8 + 2 + 3 + k / 3] = ev;
				}
			}

#ifndef _DEBUG
			Serializable::Store(
				vals,
				size * 8,
				"particleData2.txt",
				frame);
#else
			Serializable::Compare(
				vals,
				"particleData2.txt",
				frame);
#endif
			delete[] particleCPU;
			delete[] vals;
				}
#endif
	}
	/*
	std::cout << "5____________" << std::endl;
	{
		SnowParticle* particlesCPU =
			new SnowParticle[numParticles];

		hipError_t(hipMemcpy(
			particlesCPU,
			particles,
			numParticles * sizeof(SnowParticle),
			hipMemcpyDeviceToHost));

		for (uint i = 0; i < numParticles; i++)
		{
			std::cout << i << " " << std::endl;
			particlesCPU[i].plasticity.print();

			std::cout << std::endl;
		}

		system("PAUSE");
	}*/
}

void Simulation::Draw()
{
	glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
	{
		SnowParticle* particlesCPU =
			new SnowParticle[numParticles];

		hipError_t(hipMemcpy(
			particlesCPU,
			particles,
			numParticles * sizeof(SnowParticle),
			hipMemcpyDeviceToHost));

		glColor3f(0.f, 1.f, 0.f);

		{
			GLfloat lDiffuse[] =
			{ 0.f, 0.f, 0.f, 1.f };
			glMaterialfv(
				GL_FRONT,
				GL_DIFFUSE,
				lDiffuse);
		}

		for (uint i = 0; i < numParticles; i++)
		{
			glPushMatrix();
			{
				glTranslatef(
					particlesCPU[i].position.x,
					particlesCPU[i].position.y,
					particlesCPU[i].position.z);

				Cube::Render(.1f);
			}
			glPopMatrix();
		}

		glColor3f(1.f, 1.f, 1.f);

		delete[] particlesCPU;
	}

	glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
	{
		Obstacle* obstaclesCPU =
			new Obstacle[numParticles];

		hipError_t(hipMemcpy(
			obstaclesCPU,
			obstacles,
			numObstacles * sizeof(Obstacle),
			hipMemcpyDeviceToHost));

		for (uint i = 0; i < numObstacles; i++)
		{
			switch (obstaclesCPU[i].type)
			{
			case 0:
			{
				glColor4f(
					1,
					0,
					0,
					1.0f);

				{
					GLfloat lDiffuse[] =
					{	1,
						0,
						0,
						.7f };
					glMaterialfv(
						GL_FRONT,
						GL_DIFFUSE,
						lDiffuse);
				}

				glPushMatrix();
				{
					Plane::Render(
						obstaclesCPU[i].pos,
						-obstaclesCPU[i].misc,
						100.f);
				}
				glPopMatrix();

				break;
			}
			case 1:
			{
				glColor4f(1.f, 0.f, 0.f, 1.f);

				{
					GLfloat lDiffuse[] =
					{ 1.f, 0.f, 0.f, 1.f };
					glMaterialfv(
						GL_FRONT,
						GL_DIFFUSE,
						lDiffuse);
				}

				glPushMatrix();
				{
					glTranslatef(
						obstaclesCPU[i].pos.x,
						obstaclesCPU[i].pos.y,
						obstaclesCPU[i].pos.z);

					Sphere::Render(obstaclesCPU[i].misc.x);
				}
				glPopMatrix();

				break;
			}
			case 2:
			{
				glColor4f(
					fabs(obstaclesCPU[i].misc.x) * .5f + sign(obstaclesCPU[i].misc.x) * .5 + .2,
					fabs(obstaclesCPU[i].misc.y) * .5f + sign(obstaclesCPU[i].misc.y) * .5 + .2,
					fabs(obstaclesCPU[i].misc.z) * .5f + sign(obstaclesCPU[i].misc.z) * .5 + .2,
					1.0f);

				{
					GLfloat lDiffuse[] = {
						fabs(obstaclesCPU[i].misc.x) * .5f + sign(obstaclesCPU[i].misc.x) * .5 + .2,
						fabs(obstaclesCPU[i].misc.y) * .5f + sign(obstaclesCPU[i].misc.y) * .5 + .2,
						fabs(obstaclesCPU[i].misc.z) * .5f + sign(obstaclesCPU[i].misc.z) * .5 + .2,
						1.0f };
					glMaterialfv(
						GL_FRONT,
						GL_DIFFUSE,
						lDiffuse);

					GLfloat lAmbient[] = {
						fabs(obstaclesCPU[i].misc.x) + sign(obstaclesCPU[i].misc.x) * .5 + .2,
						fabs(obstaclesCPU[i].misc.y) + sign(obstaclesCPU[i].misc.y) * .5 + .2,
						fabs(obstaclesCPU[i].misc.z) + sign(obstaclesCPU[i].misc.z) * .5 + .2,
						1.0 };
					glMaterialfv(GL_FRONT, GL_AMBIENT, lAmbient);
				}

				glPushMatrix();
				{
					Plane::Render(
						obstaclesCPU[i].pos,
						obstaclesCPU[i].misc,
						10.f);
				}
				glPopMatrix();

				{
					GLfloat lAmbient[] = {
									1.0,
									1.0,
									1.0,
									1.0 };
					glMaterialfv(GL_FRONT, GL_AMBIENT, lAmbient);
				}

				break;
			}
			}
		}

		{
			glColor4f(1.f, 1.f, 1.f, 1.f);
		}
		
		delete[] obstaclesCPU;
	}
}